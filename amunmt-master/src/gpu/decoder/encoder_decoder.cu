#include <iostream>

#include "common/god.h"

#include "encoder_decoder.h"
#include "gpu/mblas/matrix_functions.h"
#include "gpu/dl4mt/dl4mt.h"
#include "gpu/decoder/encoder_decoder_state.h"
#include "gpu/decoder/best_hyps.h"

using namespace std;

namespace GPU {

////////////////////////////////////////////
std::string EncoderDecoderState::Debug() const
{
	return states_.Debug();
}

mblas::Matrix& EncoderDecoderState::GetStates() {
  return states_;
}

mblas::Matrix& EncoderDecoderState::GetEmbeddings() {
  return embeddings_;
}

const mblas::Matrix& EncoderDecoderState::GetStates() const {
  return states_;
}

const mblas::Matrix& EncoderDecoderState::GetEmbeddings() const {
  return embeddings_;
}

////////////////////////////////////////////

EncoderDecoder::EncoderDecoder(const std::string& name,
               const YAML::Node& config,
               size_t tab,
               const Weights& model)
  : Scorer(name, config, tab),
    model_(model),
    encoder_(new Encoder(model_)),
    decoder_(new Decoder(model_)),
    indeces_(God::Get<size_t>("beam-size")),
    SourceContext_(new mblas::Matrix())
{}

void EncoderDecoder::Score(const State& in,State& out) {
  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();

  decoder_->MakeStep(edOut.GetStates(),
                     edIn.GetStates(),
                     edIn.GetEmbeddings(),
                     *SourceContext_);
}

State* EncoderDecoder::NewState() {
  return new EDState();
}

void EncoderDecoder::BeginSentenceState(State& state) {
  EDState& edState = state.get<EDState>();
  decoder_->EmptyState(edState.GetStates(), *SourceContext_, 1);
  decoder_->EmptyEmbedding(edState.GetEmbeddings(), 1);
}

void EncoderDecoder::SetSource(const Sentence& source) {
  encoder_->GetContext(source.GetWords(tab_), *SourceContext_);
}

void EncoderDecoder::AssembleBeamState(const State& in,
                               const Beam& beam,
                               State& out) {
  std::vector<size_t> beamWords;
  std::vector<size_t> beamStateIds;
  for(auto h : beam) {
     beamWords.push_back(h->GetWord());
     beamStateIds.push_back(h->GetPrevStateIndex());
  }

  const EDState& edIn = in.get<EDState>();
  EDState& edOut = out.get<EDState>();
  indeces_.resize(beamStateIds.size());
  thrust::host_vector<size_t> tmp = beamStateIds;
  mblas::copy_n(tmp.begin(), beamStateIds.size(), indeces_.begin());

  mblas::Assemble(edOut.GetStates(), edIn.GetStates(), indeces_);
  decoder_->Lookup(edOut.GetEmbeddings(), beamWords);
}

void EncoderDecoder::GetAttention(mblas::Matrix& Attention) {
  decoder_->GetAttention(Attention);
}

BaseMatrix& EncoderDecoder::GetProbs() {
  return decoder_->GetProbs();
}

mblas::Matrix& EncoderDecoder::GetAttention() {
  return decoder_->GetAttention();
}

size_t EncoderDecoder::GetVocabSize() const {
  return decoder_->GetVocabSize();
}

void EncoderDecoder::Filter(const std::vector<size_t>& filterIds) {
  decoder_->Filter(filterIds);
}

EncoderDecoder::~EncoderDecoder() {}

////////////////////////////////////////////
EncoderDecoderLoader::EncoderDecoderLoader(const std::string name,
                     const YAML::Node& config)
 : Loader(name, config) {}

void EncoderDecoderLoader::Load() {
  std::string path = Get<std::string>("path");
  auto devices = God::Get<std::vector<size_t>>("devices");
  ThreadPool devicePool(devices.size());
  weights_.resize(devices.size());

  size_t i = 0;
  for(auto d : devices) {
    devicePool.enqueue([i, d, &path, this] {
      LOG(info) << "Loading model " << path << " onto gpu" << d;
      hipSetDevice(d);
      weights_[i].reset(new Weights(path, d));
    });
    ++i;
  }
}

ScorerPtr EncoderDecoderLoader::NewScorer(size_t taskId) {
  size_t i = taskId % weights_.size();
  size_t d = weights_[i]->GetDevice();
  hipSetDevice(d);
  size_t tab = Has("tab") ? Get<size_t>("tab") : 0;
  return ScorerPtr(new EncoderDecoder(name_, config_,
                                      tab, *weights_[i]));
}

BestHypsType EncoderDecoderLoader::GetBestHyps() {
  return GPU::BestHyps();
}

}

