#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <string>
#include <boost/timer/timer.hpp>

#include "nmt.h"
#include "mblas/matrix.h"
#include "dl4mt.h"
#include "common/vocab.h"
#include "common/states.h"


using namespace mblas;

NMT::NMT(const boost::shared_ptr<Weights> model,
         const boost::shared_ptr<Vocab> src,
         const boost::shared_ptr<Vocab> trg)
  : debug_(false), w_(model), src_(src), trg_(trg),
    encoder_(new Encoder(*w_)), decoder_(new Decoder(*w_)),
    states_(new States()), firstWord_(true)
  {
    for(size_t i = 0; i < trg_->size(); ++i)
      filteredId_.push_back(i);
  }
  
void NMT::PrintState(StateInfoPtr ptr) {
  std::cerr << *ptr << std::endl;
}

size_t NMT::GetDevices(size_t maxDevices) {
  int num_gpus = 0;   // number of CUDA GPUs
  hipGetDeviceCount(&num_gpus);
  std::cerr << "Number of CUDA devices: " << num_gpus << std::endl;
  
  for (int i = 0; i < num_gpus; i++) {
      hipDeviceProp_t dprop;
      hipGetDeviceProperties(&dprop, i);
      std::cerr << i << ": " << dprop.name << std::endl;
  }
  return (size_t)std::min(num_gpus, (int)maxDevices);
}

void NMT::SetDevice() {
  hipSetDevice(w_->GetDevice());
  CublasHandler::StaticHandle();
}


size_t NMT::GetDevice() {
  return w_->GetDevice();
}

void NMT::ClearStates() { 
  firstWord_ = true;
  states_->Clear();
}

boost::shared_ptr<Weights> NMT::NewModel(const std::string& path, size_t device) {
  std::cerr << "Got device " << device << std::endl;
  hipSetDevice(device);
  CublasHandler::StaticHandle();
  boost::shared_ptr<Weights> weights(new Weights(path, device));
  return weights;
}

boost::shared_ptr<Vocab> NMT::NewVocab(const std::string& path) {
  boost::shared_ptr<Vocab> vocab(new Vocab(path));
  return vocab;
}

size_t NMT::TargetVocab(const std::string& str) {
  return (*trg_)[str];
}

void NMT::CalcSourceContext(const std::vector<std::string>& s) {  
  std::vector<size_t> words(s.size());
  std::transform(s.begin(), s.end(), words.begin(),
                 [&](const std::string& w) { return (*src_)[w]; });
  words.push_back((*src_)["</s>"]);
  
  SourceContext_.reset(new Matrix());
  Matrix& SC = *boost::static_pointer_cast<Matrix>(SourceContext_);
  encoder_->GetContext(words, SC);
}

StateInfoPtr NMT::EmptyState() {
  Matrix& SC = *boost::static_pointer_cast<Matrix>(SourceContext_);
  Matrix Empty;
  decoder_->EmptyState(Empty, SC, 1);
  std::vector<StateInfoPtr> infos;
  states_->SaveStates(infos, Empty);
  return infos.back();
}

void NMT::FilterTargetVocab(const std::set<std::string>& filter) {
  filteredId_.clear();
  filteredId_.resize(trg_->size(), 1); // set all to UNK
  
  std::vector<size_t> numericFilter;
  size_t k = 0;
  for(auto& s : filter) {
    size_t id = (*trg_)[s];
    numericFilter.push_back(id);
    filteredId_[id] = k;
    k++;
  }
  // eol
  numericFilter.push_back(numericFilter.size());
  decoder_->Filter(numericFilter);
}

void NMT::BatchSteps(const Batches& batches, LastWords& lastWords,
                     Scores& probsOut, Scores& unksOut, StateInfos& stateInfos,
                     bool firstWord) {
  Matrix& sourceContext = *boost::static_pointer_cast<Matrix>(SourceContext_);

  Matrix prevEmbeddings;
  Matrix nextEmbeddings;
  Matrix prevStates;
  Matrix probs;
  Matrix nextStates;

  if(firstWord) {
    decoder_->EmptyEmbedding(prevEmbeddings, lastWords.size());
  }
  else {
    // Not the first word
    decoder_->Lookup(prevEmbeddings, lastWords);
  }

  states_->ConstructStates(prevStates, stateInfos);

  for(auto& batch : batches) {
    decoder_->MakeStep(nextStates, nextEmbeddings, probs,
                       batch, prevStates, prevEmbeddings, sourceContext);

    StateInfos tempStates;
    states_->SaveStates(tempStates, nextStates);

    for(size_t i = 0; i < batch.size(); ++i) {
      if(batch[i] != 0) {
        float p = probs(i, filteredId_[batch[i]]);
        probsOut[i] += log(p);
        stateInfos[i] = tempStates[i];
      }
      if(batch[i] == 1) {
        unksOut[i]++;
      }
    }
    Swap(nextStates, prevStates);
    Swap(nextEmbeddings, prevEmbeddings);
  }
}

void NMT::OnePhrase(
  const std::vector<std::string>& phrase,
  const std::string& lastWord,
  bool firstWord,
  StateInfoPtr inputState,
  float& prob, size_t& unks,
  StateInfoPtr& outputState) {
  
  Matrix& sourceContext = *boost::static_pointer_cast<Matrix>(SourceContext_);
  
  Matrix prevEmbeddings;
  Matrix nextEmbeddings;
  Matrix prevStates;
  Matrix probs;
  Matrix alignedSourceContext;
  Matrix nextStates;
    
  if(firstWord) {
    decoder_->EmptyEmbedding(prevEmbeddings, 1);
  }
  else {
    // Not the first word
    std::vector<size_t> ids = { (*trg_)[lastWord] };
    decoder_->Lookup(prevEmbeddings, ids);
  }
    
  std::vector<StateInfoPtr> inputStates = { inputState };
  states_->ConstructStates(prevStates, inputStates);
    
  for(auto& w : phrase) {
    size_t id = (*trg_)[w];
    std::vector<size_t> nextIds = { id };
    if(id == 1)
      unks++;
    
    decoder_->MakeStep(nextStates, nextEmbeddings, probs,
                       nextIds, prevStates, prevEmbeddings, sourceContext);
    
    float p = probs(0, filteredId_[id]);
    prob += log(p);
    
    Swap(nextStates, prevStates);
    Swap(nextEmbeddings, prevEmbeddings);
  }
  
  std::vector<StateInfoPtr> outputStates;
  states_->SaveStates(outputStates, prevStates);
  outputState = outputStates.back();
}

void NMT::MakeStep(
  const std::vector<std::string>& nextWords,
  const std::vector<std::string>& lastWords,
  std::vector<StateInfoPtr>& inputStates,
  std::vector<double>& logProbs,
  std::vector<StateInfoPtr>& outputStates,
  std::vector<bool>& unks) {
  
  Matrix& sourceContext = *boost::static_pointer_cast<Matrix>(SourceContext_);
  
  Matrix lastEmbeddings;
  if(firstWord_) {
    firstWord_ = false;
    // Only empty state in state cache, so this is the first word
    decoder_->EmptyEmbedding(lastEmbeddings, lastWords.size());
  }
  else {
    // Not the first word
    std::vector<size_t> lastIds(lastWords.size());
    std::transform(lastWords.begin(), lastWords.end(), lastIds.begin(),
                   [&](const std::string& w) { return (*trg_)[w]; });
    decoder_->Lookup(lastEmbeddings, lastIds);
  }
  
  Matrix nextEmbeddings;
  std::vector<size_t> nextIds(nextWords.size());
  std::transform(nextWords.begin(), nextWords.end(), nextIds.begin(),
                 [&](const std::string& w) { return (*trg_)[w]; });
  
  Matrix prevStates;
  states_->ConstructStates(prevStates, inputStates);

  Matrix probs;
  Matrix nextStates;
  
  decoder_->MakeStep(nextStates, nextEmbeddings, probs,
                     nextIds, prevStates, lastEmbeddings, sourceContext);
  
  states_->SaveStates(outputStates, nextStates);
  
  for(auto id : nextIds) {
    if(id != 1)
      unks.push_back(true);
    else
      unks.push_back(false);
  }
  
  for(size_t i = 0; i < nextIds.size(); ++i) {
    float p = probs(i, filteredId_[nextIds[i]]);
    //float p = probs(i, nextIds[i]);
    logProbs.push_back(log(p));
  }
  
}
